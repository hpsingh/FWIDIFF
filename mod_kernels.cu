#include "hip/hip_runtime.h"
__global__ void cuda_set_sg(int *sxz, int sxbeg, int szbeg, int jsx, int jsz, int ns, int nz)
/*< set the positions of sources/geophones >*/
{
	int id=blockDim.x*blockIdx.x + threadIdx.x;
    	if (id<ns) sxz[id]=(szbeg+id*jsz)+nz*(sxbeg+id*jsx);
}

__global__ void cuda_ricker_wavelet(float *wlt, float amp, float fm, float dt, int nt)
/*< generate ricker wavelet with time deley >*/
{
	int it=blockDim.x*blockIdx.x + threadIdx.x;
    	if (it<nt)
	{
	    	float tmp = PI*fm*(it*dt-1.0/fm);
	    	tmp *=tmp;
		wlt[it]=amp*(1.0-2.0*tmp)*expf(-tmp);
	}
}

__global__ void cuda_add_source(float *p, float *source, int *sxz, int ns, bool add)
/*< add==true, add (inject) the source; add==false, subtract the source >*/
{
	int id=blockDim.x*blockIdx.x + threadIdx.x;
    	if (id<ns)
	{
		if (add)	p[sxz[id]]+=source[id];
		else 		p[sxz[id]]-=source[id];
	}	
}

__global__ void cuda_record(float*p, float *seis, int *gxz, int ng)
/*< record the seismogram at time it >*/
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<ng) seis[id]=p[gxz[id]];
}

__global__ void cuda_step_forward(float *p0, float *p1, float *vv, float dtz, float dtx, int nz, int nx)
/*< step forward: dtz=dt/dx; dtx=dt/dz; >*/
{
	int i1=threadIdx.x+blockIdx.x*blockDim.x;
	int i2=threadIdx.y+blockIdx.y*blockDim.y;
	int id=i1+i2*nz;

	__shared__ float s_p0[Block_Size2+2][Block_Size1+2];
	__shared__ float s_p1[Block_Size2+2][Block_Size1+2];
	if(threadIdx.x<1)
	{
		s_p0[threadIdx.y+1][threadIdx.x]=(blockIdx.x>0)?p0[id-1]:0.0;	
		s_p1[threadIdx.y+1][threadIdx.x]=(blockIdx.x>0)?p1[id-1]:0.0;
	}
	if(threadIdx.x>=blockDim.x-1)
	{
		s_p0[threadIdx.y+1][threadIdx.x+2]=(blockIdx.x<gridDim.x-1)?p0[id+1]:0.0;
		s_p1[threadIdx.y+1][threadIdx.x+2]=(blockIdx.x<gridDim.x-1)?p1[id+1]:0.0;
	}
	if(threadIdx.y<1)
	{
		s_p0[threadIdx.y][threadIdx.x+1]=(blockIdx.y>0)?p1[id-nz]:0.0;
	 	s_p1[threadIdx.y][threadIdx.x+1]=(blockIdx.y>0)?p1[id-nz]:0.0;
	}
	if(threadIdx.y>=blockDim.y-1)
	{
		s_p0[threadIdx.y+2][threadIdx.x+1]=(blockIdx.y<gridDim.y-1)?p1[id+nz]:0.0;
		s_p1[threadIdx.y+2][threadIdx.x+1]=(blockIdx.y<gridDim.y-1)?p1[id+nz]:0.0;
	}
	s_p0[threadIdx.y+1][threadIdx.x+1]=p0[id];
	s_p1[threadIdx.y+1][threadIdx.x+1]=p1[id];
	__syncthreads();

	if (i1<nz && i2<nx)
	{
		float v1=vv[id]*dtz;
		float v2=vv[id]*dtx; 
		float c1=v1*v1*(s_p1[threadIdx.y+1][threadIdx.x+2]-2.0*s_p1[threadIdx.y+1][threadIdx.x+1]+s_p1[threadIdx.y+1][threadIdx.x]);
		float c2=v2*v2*(s_p1[threadIdx.y+2][threadIdx.x+1]-2.0*s_p1[threadIdx.y+1][threadIdx.x+1]+s_p1[threadIdx.y][threadIdx.x+1]);
/*
		if(i1==0)// top boundary is free surface boundary condition, commentted!!
		{
			c1=v1*(-s_p1[threadIdx.y+1][threadIdx.x+1]+s_p1[threadIdx.y+1][threadIdx.x+2]
						+s_p0[threadIdx.y+1][threadIdx.x+1]-s_p0[threadIdx.y+1][threadIdx.x+2]);
			if(i2>0 && i2<nx-1) c2=0.5*c2;
		}
*/
		if(i1==nz-1) /* bottom boundary */
		{
			c1=v1*(s_p1[threadIdx.y+1][threadIdx.x]-s_p1[threadIdx.y+1][threadIdx.x+1]
						-s_p0[threadIdx.y+1][threadIdx.x]+s_p0[threadIdx.y+1][threadIdx.x+1]);
			if(i2>0 && i2<nx-1) c2=0.5*c2;
		}

		if(i2==0)/* left boundary */
		{
			if(i1>0 && i1<nz-1) c1=0.5*c1;
			c2=v2*(-s_p1[threadIdx.y+1][threadIdx.x+1]+s_p1[threadIdx.y+2][threadIdx.x+1]
						+s_p0[threadIdx.y+1][threadIdx.x+1]-s_p0[threadIdx.y+2][threadIdx.x+1]);

		}

		if(i2==nx-1) /* right boundary */
		{
			if(i1>0 && i1<nz-1) c1=0.5*c1;
			c2=v2*(s_p1[threadIdx.y][threadIdx.x+1]-s_p1[threadIdx.y+1][threadIdx.x+1]
						-s_p0[threadIdx.y][threadIdx.x+1]+s_p0[threadIdx.y+1][threadIdx.x+1]);
		}
		p0[id]=2.0*s_p1[threadIdx.y+1][threadIdx.x+1]-s_p0[threadIdx.y+1][threadIdx.x+1]+c1+c2;
	}
}
