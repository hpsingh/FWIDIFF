#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

extern "C" {
#include <rsf.h>
}

#ifndef true
#define true    (1)
#endif
#ifndef false
#define false   (0)
#endif
#ifndef EPS
#define EPS	SF_EPS
#endif

#define PI 	SF_PI
#define Block_Size1 16	/* 1st dim block size */
#define Block_Size2 16  /* 2nd dim block size */
#define Block_Size  512	/* vector computation blocklength */
#define nbell	2	/* radius of Gaussian bell: diameter=2*nbell+1 */

#include "mod_kernels.cu"

void sf_check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}

void matrix_transpose(float *matrix, float *trans, int n1, int n2)
/*< matrix transpose: matrix tansposed to be trans >*/
{
	int i1, i2;

	for(i2=0; i2<n2; i2++)
	for(i1=0; i1<n1; i1++)
	    trans[i2+n2*i1]=matrix[i1+n1*i2];
}

void expand(float*vv, float *v0, int nz, int nx, int nz1, int nx1)
/*< round up the model size to be multiples of block size >*/
{
	int i1,i2,i11,i22;

	for(i2=0; i2<nx; i2++)
	for(i1=0; i1<nz; i1++)
	{
		i11=(i1<nz1)?i1:(nz1-1);
		i22=(i2<nx1)?i2:(nx1-1);
		vv[i1+i2*nz]=v0[i11+nz1*i22];
	}	
}


void window(float *v0,float *vv, int nz, int nx, int nz1, int nx1)
/*< window the portion to be the same size as initial model >*/
{
	int i1, i2;

	for(i2=0; i2<nx1; i2++)
	for(i1=0; i1<nz1; i1++)
		  v0[i1+i2*nz1]=vv[i1+nz*i2];
}


int main(int argc, char *argv[])
{
	/* variables on host */
	bool csdgather, chk;
	int nz, nx, nz1, nx1, nt, ns, ng;
	int is, it,kt, distx, distz;
	int sxbeg,szbeg,gxbeg,gzbeg,jsx,jsz,jgx,jgz;
	float dx, dz, fm, dt, dtx, dtz, mstimer, amp, totaltime=0;
	float *v0, *dobs, *vv, *trans, *ptr=NULL;
	/* variables on device */
	int 	*d_sxz, *d_gxz;			
	float 	*d_wlt, *d_vv, *d_sp0, *d_sp1, *d_dobs;
	sf_file vinit, shots, check=NULL, time;

    	/* initialize Madagascar */
    	sf_init(argc,argv);

    	/*< set up I/O files >*/
    	vinit=sf_input ("in");   /* initial velocity model, unit=m/s */
    	shots=sf_output("out");  /* output image with correlation imaging condition */ 
	time=sf_output("time");  /* output total time */

    	/* get parameters for forward modeling */
    	if (!sf_histint(vinit,"n1",&nz1)) sf_error("no n1");/* n1 */
    	if (!sf_histint(vinit,"n2",&nx1)) sf_error("no n2");/* n2 */
    	if (!sf_histfloat(vinit,"d1",&dz)) sf_error("no d1");/* d1 */
   	if (!sf_histfloat(vinit,"d2",&dx)) sf_error("no d2");/* d2 */

    	if(!sf_getbool("chk",&chk)) chk=false;
    	/*check whether GPU-CPU implementation coincide with each other or not */
	if(chk){
    		if (!sf_getint("kt",&kt))  kt=100;/* check it at it=100 */ 
		check=sf_output("check");/* check reconstructed shotsnap */
	}
	if (!sf_getfloat("amp",&amp)) amp=1000;
	/* maximum amplitude of ricker */
    	if (!sf_getfloat("fm",&fm)) fm=10;	
	/* dominant freq of ricker */
    	if (!sf_getfloat("dt",&dt)) sf_error("no dt");	
	/* time interval */
    	if (!sf_getint("nt",&nt))   sf_error("no nt");	
	/* total modeling time steps */
    	if (!sf_getint("ns",&ns))   sf_error("no ns");	
	/* total shots */
    	if (!sf_getint("ng",&ng))   sf_error("no ng");	
	/* total receivers in each shot */	
    	if (!sf_getint("jsx",&jsx))   sf_error("no jsx");
	/* source x-axis  jump interval  */
    	if (!sf_getint("jsz",&jsz))   jsz=0;
	/* source z-axis jump interval  */
    	if (!sf_getint("jgx",&jgx))   jgx=1;
	/* receiver x-axis jump interval */
    	if (!sf_getint("jgz",&jgz))   jgz=0;
	/* receiver z-axis jump interval */
    	if (!sf_getint("sxbeg",&sxbeg))   sf_error("no sxbeg");
	/* x-begining index of sources, starting from 0 */
    	if (!sf_getint("szbeg",&szbeg))   sf_error("no szbeg");
	/* z-begining index of sources, starting from 0 */
    	if (!sf_getint("gxbeg",&gxbeg))   sf_error("no gxbeg");
	/* x-begining index of receivers, starting from 0 */
    	if (!sf_getint("gzbeg",&gzbeg))   sf_error("no gzbeg");
	/* z-begining index of receivers, starting from 0 */
	if (!sf_getbool("csdgather",&csdgather)) csdgather=false;
	/* default, common shot-gather; if n, record at every point*/

	/* put the labels, legends and parameters in output */
	sf_putint(shots,"n1",nt);	
	sf_putint(shots,"n2",ng);
	sf_putint(shots,"n3",ns);
	sf_putfloat(shots,"d1",dt);
	sf_putfloat(shots,"d2",jgx*dx);
	sf_putfloat(shots,"o1",0);
	sf_putstring(shots,"label1","Time");
	sf_putstring(shots,"label2","Lateral");
	sf_putstring(shots,"label3","Shot");
	sf_putstring(shots,"unit1","sec");
	sf_putstring(shots,"unit2","m");
	sf_putfloat(shots,"amp",amp);
	sf_putfloat(shots,"fm",fm);
	sf_putint(shots,"ng",ng);
	sf_putint(shots,"szbeg",szbeg);
	sf_putint(shots,"sxbeg",sxbeg);
	sf_putint(shots,"gzbeg",gzbeg);
	sf_putint(shots,"gxbeg",gxbeg);
	sf_putint(shots,"jsx",jsx);
	sf_putint(shots,"jsz",jsz);
	sf_putint(shots,"jgx",jgx);
	sf_putint(shots,"jgz",jgz);
	sf_putint(shots,"csdgather",csdgather?1:0);
	sf_putint(time,"n1",1);
	sf_putint(time,"n2",1);

	dtx=dt/dx; 
	dtz=dt/dz; 
	/* round the size up to multiples of Block size */
	nx=(int)((nx1+Block_Size1-1)/Block_Size1)*Block_Size1;
	nz=(int)((nz1+Block_Size2-1)/Block_Size2)*Block_Size2;

	/* allocate memory for variables on host */
	v0=(float*)malloc(nz1*nx1*sizeof(float));
	vv=(float*)malloc(nz*nx*sizeof(float));
	dobs=(float*)malloc(ng*nt*sizeof(float));
	trans=(float*)malloc(ng*nt*sizeof(float));
	sf_floatread(v0,nz1*nx1,vinit);
	expand(vv, v0, nz, nx, nz1, nx1);
	memset(dobs,0,ng*nt*sizeof(float));
	memset(trans,0,ng*nt*sizeof(float));

    	hipSetDevice(0);
	sf_check_gpu_error("Failed to initialize device!");
	/* allocate memory for variables on device */
	hipMalloc(&d_vv, nz*nx*sizeof(float));
	hipMalloc(&d_sp0, nz*nx*sizeof(float));
	hipMalloc(&d_sp1, nz*nx*sizeof(float));
	hipMalloc(&d_wlt, nt*sizeof(float));
	hipMalloc(&d_sxz, nt*sizeof(float));
	hipMalloc(&d_gxz, ng*sizeof(float));
	hipMalloc(&d_dobs, ng*nt*sizeof(float));
	sf_check_gpu_error("Failed to allocate required memory!");

	/* set GPU block size */
	dim3 dimg=dim3(nz/Block_Size1, nx/Block_Size2),dimb=dim3(Block_Size1, Block_Size2); 

	hipMemcpy(d_vv, vv, nz*nx*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(d_sp0,0,nz*nx*sizeof(float));
	hipMemset(d_sp1,0,nz*nx*sizeof(float));
	cuda_ricker_wavelet<<<(nt+511)/512,512>>>(d_wlt, amp, fm, dt, nt);
	/* configure source/geophone geometry */
	if (!(sxbeg>=0 && szbeg>=0 && sxbeg+(ns-1)*jsx<nx1 && szbeg+(ns-1)*jsz<nz1))	
	{ sf_error("sources exceeds the computing zone!\n"); exit(1);}
	cuda_set_sg<<<(ns+511)/512,512>>>(d_sxz, sxbeg, szbeg, jsx, jsz, ns, nz);
	distx=sxbeg-gxbeg;
	distz=szbeg-gzbeg;
	if (!(gxbeg>=0 && gzbeg>=0 && gxbeg+(ng-1)*jgx<nx1 && gzbeg+(ng-1)*jgz<nz1))	
	{ sf_error("geophones exceeds the computing zone!\n"); exit(1);}
	if (csdgather)	{
		if (!(	(sxbeg+(ns-1)*jsx)+(ng-1)*jgx-distx <nx1  && (szbeg+(ns-1)*jsz)+(ng-1)*jgz-distz <nz1))	
		{ sf_error("geophones exceeds the computing zone!\n"); exit(1);}
	}
	cuda_set_sg<<<(ng+511)/512,512>>>(d_gxz, gxbeg, gzbeg, jgx, jgz, ng, nz);

	/* creat cuda timer */
	hipEvent_t start, stop;
  	hipEventCreate(&start);	
	hipEventCreate(&stop);
	for(is=0;is<ns;is++)/* generate ns shots one by one */
	{
		hipEventRecord(start);
		hipMemset(d_dobs, 0, ng*nt*sizeof(float));
		if (csdgather)	{/* reset position according to gather type */
			gxbeg=sxbeg+is*jsx-distx;
			cuda_set_sg<<<(ng+511)/512, 512>>>(d_gxz, gxbeg, gzbeg, jgx, jgz, ng, nz);
		}
		hipMemset(d_sp0, 0, nz*nx*sizeof(float));
		hipMemset(d_sp1, 0, nz*nx*sizeof(float));
		/* forward modeling */
		for(it=0; it<nt; it++)
		{
			cuda_add_source<<<1,1>>>(d_sp1, &d_wlt[it], &d_sxz[is], 1, true);
			cuda_step_forward<<<dimg,dimb>>>(d_sp0, d_sp1, d_vv, dtz, dtx, nz, nx);
			ptr=d_sp0; d_sp0=d_sp1; d_sp1=ptr;
			cuda_record<<<(ng+511)/512, 512>>>(d_sp0, &d_dobs[it*ng], d_gxz, ng);

			if(chk && it==kt){/* record a snapshot */			
				float *test=(float*)malloc(nz*nx*sizeof(float));

				hipMemcpy(test, d_sp0, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
				window(v0, test, nz, nx, nz1, nx1);
				sf_floatwrite(v0,nz*nx, check);
				
				free(test);
			}
		}
		/* save the modeled shot in trace-by-trace format */
		hipMemcpy(dobs, d_dobs, ng*nt*sizeof(float), hipMemcpyDeviceToHost);
		matrix_transpose(dobs, trans, ng, nt);
		sf_floatwrite(trans,ng*nt,shots);

		hipEventRecord(stop);
  		hipEventSynchronize(stop);
  		hipEventElapsedTime(&mstimer, start, stop);
    		sf_warning("shot %d finished: %f (s)",is+1, mstimer*1.e-3);
		totaltime+=mstimer*1.e-3;/* mstimer with different unit, be careful! */
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	totaltime/=ns;/* compute the average time cost */
	sf_floatwrite(&totaltime,1,time);

	/* free host variables */
	free(v0);
	free(vv);
	free(dobs);
	free(trans);
	/* free device variables */
	hipFree(d_vv);
	hipFree(d_sp0);
	hipFree(d_sp1);
	hipFree(d_wlt);
	hipFree(d_sxz);
	hipFree(d_gxz);
	hipFree(d_dobs);
	sf_check_gpu_error("Failed to free the allocated memory!");

	return 0;
}
