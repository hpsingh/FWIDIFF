#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

extern "C" {
#include <rsf.h>
}

#ifndef true
#define true    (1)
#endif
#ifndef false
#define false   (0)
#endif
#ifndef EPS
#define EPS	SF_EPS
#endif

#define PI 	SF_PI
#define Block_Size1 16	/* 1st dim block size */
#define Block_Size2 16	/* 2nd dim block size */
#define Block_Size  512	/* vector computation blocklength */

#include "mod_kernel.cu"

void sf_check_gpu_error (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}

void expand(float*vv, float *v0, int nz, int nx, int nz1, int nx1)
/*< round up the model size to be multiples of block size >*/
{
	int i1,i2,i11,i22;

	for(i2=0; i2<nx; i2++)
	for(i1=0; i1<nz; i1++)
	{
		i11=(i1<nz1)?i1:(nz1-1);
		i22=(i2<nx1)?i2:(nx1-1);
		vv[i1+i2*nz]=v0[i11+nz1*i22];
	}	
}

void window(float *v0,float *vv, int nz, int nx, int nz1, int nx1)
/*< window the portion to be the same size as initial model >*/
{
	int i1, i2;

	for(i2=0; i2<nx1; i2++)
	for(i1=0; i1<nz1; i1++)
		v0[i1+i2*nz1]=vv[i1+nz*i2];
}

void matrix_transpose(float *matrix, float *trans, int n1, int n2)
/*< matrix transpose: matrix tansposed to be trans >*/
{
	int i1, i2;

	for(i2=0; i2<n2; i2++)
	for(i1=0; i1<n1; i1++)
	    trans[i2+n2*i1]=matrix[i1+n1*i2];
}

int main(int argc, char *argv[])
{
	/* variables on host */
	bool verb, precon, csdgather;
	int is, it, iter, niter, distx, distz, csd, rbell;
	int nz, nx, nz1, nx1, nt, ns, ng;
	int sxbeg, szbeg, gxbeg, gzbeg, jsx, jsz, jgx, jgz;/*  parameters of acquisition geometery */
	float dx, dz, fm, dt, dtx, dtz, mstimer,amp, obj1, obj, beta, epsil, alpha;
	float *v0, *vv, *dobs, *trans, *objval, *ptr=NULL;
	sf_file vinit, shots, vupdates, grads, objs, illums;

	/* variables on device */
	int 	*d_sxz, *d_gxz;			
	float 	*d_wlt, *d_vv, *d_illum, *d_lap, *d_vtmp, *d_sp0, *d_sp1, *d_gp0, *d_gp1,*d_bndr;
	float	*d_dobs, *d_dcal, *d_derr, *d_g0, *d_g1, *d_cg, *d_pars, *d_alpha1, *d_alpha2;

    	/* initialize Madagascar */
    	sf_init(argc,argv);

    	/* set up I/O files */
    	vinit=sf_input ("in");   /* initial velocity model, unit=m/s */
	shots=sf_input("shots"); /* recorded shots from exact velocity model */
    	vupdates=sf_output("out"); /* updated velocity in iterations */ 
    	grads=sf_output("grads");  /* gradient in iterations */ 
	objs=sf_output("objs");/* values of objective function in iterations */
	illums=sf_output("illums");/* source illumination in iterations */

    	/* get parameters from velocity model and recorded shots */
	if (!sf_getbool("verb",&verb)) verb=true;/* vebosity */
    	if (!sf_histint(vinit,"n1",&nz1)) sf_error("no n1");/* n1 */
    	if (!sf_histint(vinit,"n2",&nx1)) sf_error("no n2");/* n2 */
    	if (!sf_histfloat(vinit,"d1",&dz)) sf_error("no d1");/* d1 */
   	if (!sf_histfloat(vinit,"d2",&dx)) sf_error("no d2");/* d2 */
	if (!sf_getbool("precon",&precon)) precon=false;/* precondition or not */
    	if (!sf_getint("niter",&niter))   niter=100;	/* number of iterations */
	if (!sf_getint("rbell",&rbell))	  rbell=2;	/* radius of bell smooth */

   	if (!sf_histint(shots,"n1",&nt)) sf_error("no nt");
	/* total modeling time steps */
   	if (!sf_histint(shots,"n2",&ng)) sf_error("no ng");
	/* total receivers in each shot */
   	if (!sf_histint(shots,"n3",&ns)) sf_error("no ns");
	/* number of shots */
   	if (!sf_histfloat(shots,"d1",&dt)) sf_error("no dt");
	/* time sampling interval */
   	if (!sf_histfloat(shots,"amp",&amp)) sf_error("no amp");
	/* maximum amplitude of ricker */
   	if (!sf_histfloat(shots,"fm",&fm)) sf_error("no fm");
	/* dominant freq of ricker */
   	if (!sf_histint(shots,"sxbeg",&sxbeg)) sf_error("no sxbeg");
	/* x-begining index of sources, starting from 0 */
   	if (!sf_histint(shots,"szbeg",&szbeg)) sf_error("no szbeg");
	/* x-begining index of sources, starting from 0 */
   	if (!sf_histint(shots,"gxbeg",&gxbeg)) sf_error("no gxbeg");
	/* x-begining index of receivers, starting from 0 */
   	if (!sf_histint(shots,"gzbeg",&gzbeg)) sf_error("no gzbeg");
	/* x-begining index of receivers, starting from 0 */
   	if (!sf_histint(shots,"jsx",&jsx)) sf_error("no jsx");
	/* source x-axis  jump interval  */
   	if (!sf_histint(shots,"jsz",&jsz)) sf_error("no jsz");
	/* source z-axis jump interval  */
   	if (!sf_histint(shots,"jgx",&jgx)) sf_error("no jgx");
	/* receiver x-axis jump interval  */
   	if (!sf_histint(shots,"jgz",&jgz)) sf_error("no jgz");
	/* receiver z-axis jump interval  */
   	if (!sf_histint(shots,"csdgather",&csd)) sf_error("csdgather or not required");
	/* default, common shot-gather; if n, record at every point*/

	/* put the labels, legends and parameters for output rsf file */
	sf_putint(vupdates,"n1",nz1);	
	sf_putint(vupdates,"n2",nx1);
	sf_putfloat(vupdates,"d1",dz);
	sf_putfloat(vupdates,"d2",dx);
	sf_putstring(vupdates,"label1","Depth");
	sf_putstring(vupdates,"label2","Distance");
	sf_putstring(vupdates,"label3","Iteration");
	sf_putint(vupdates,"n3",niter);
	sf_putint(vupdates,"d3",1);
	sf_putint(vupdates,"o3",1);
	sf_putint(grads,"n1",nz1);	
	sf_putint(grads,"n2",nx1);
	sf_putint(grads,"n3",niter);
	sf_putfloat(grads,"d1",dz);
	sf_putfloat(grads,"d2",dx);
	sf_putint(grads,"d3",1);
	sf_putint(grads,"o3",1);
	sf_putstring(grads,"label1","Depth");
	sf_putstring(grads,"label2","Distance");
	sf_putstring(grads,"label3","Iteration");
	sf_putint(illums,"n1",nz1);	
	sf_putint(illums,"n2",nx1);
	sf_putfloat(illums,"d1",dz);
	sf_putfloat(illums,"d2",dx);
	sf_putint(illums,"n3",niter);
	sf_putint(illums,"d3",1);
	sf_putint(illums,"o3",1);
	sf_putint(objs,"n1",niter);
	sf_putint(objs,"n2",1);
	sf_putint(objs,"d1",1);
	sf_putint(objs,"o1",1);

	dtx=dt/dx; 
	dtz=dt/dz; 
	csdgather=(csd>0)?true:false;
	/* round the size up to multiples of Block size */
	nx=(int)((nx1+Block_Size1-1)/Block_Size1)*Block_Size1;
	nz=(int)((nz1+Block_Size2-1)/Block_Size2)*Block_Size2; 
	dim3 dimg=dim3(nz/Block_Size1, nx/Block_Size2), dimb=dim3(Block_Size1, Block_Size2); 

	v0=(float*)malloc(nz1*nx1*sizeof(float));/* initial velocity model */
	vv=(float*)malloc(nz*nx*sizeof(float));	 /* extended velocity model, size=multiple of 16x16 block */
	dobs=(float*)malloc(ng*nt*sizeof(float));/* observations, one shot */
	trans=(float*)malloc(ng*nt*sizeof(float));/* transposed one shot */
	objval=(float*)malloc(niter*sizeof(float));/* objective/misfit function */
	sf_floatread(v0, nz1*nx1, vinit);	/* read the initial velcity model, size=nz1*nx1 */
	expand(vv, v0, nz, nx, nz1, nx1);	/* expand the model to be of size nz*nx */
	memset(dobs, 0, ng*nt*sizeof(float));	
	memset(objval, 0, niter*sizeof(float));

    	hipSetDevice(0);
	sf_check_gpu_error("Failed to initialize device!");
	/* allocate memory for device variables */
	hipMalloc(&d_vv, nz*nx*sizeof(float));	/* velocity */
	hipMalloc(&d_sp0, nz*nx*sizeof(float));/* source wavefield p0 */
	hipMalloc(&d_sp1, nz*nx*sizeof(float));/* source wavefield p1 */
	hipMalloc(&d_gp0, nz*nx*sizeof(float));/* geophone/receiver wavefield p0 */
	hipMalloc(&d_gp1, nz*nx*sizeof(float));/* geophone/receiver wavefield p1 */
	hipMalloc(&d_wlt, nt*sizeof(float));	/* ricker wavelet */
	hipMalloc(&d_sxz, ns*sizeof(float));	/* source positions */
	hipMalloc(&d_gxz, ng*sizeof(float));	/* geophone positions */
	hipMalloc(&d_bndr, nt*(2*nz+nx)*sizeof(float));/* boundaries for wavefield reconstruction */
	hipMalloc(&d_dobs, ng*nt*sizeof(float));/* observed seismic data */
	hipMalloc(&d_dcal, ng*sizeof(float));	/* calculated/synthetic seismic data */
	hipMalloc(&d_derr, ns*ng*nt*sizeof(float));/* residual/error between synthetic and observation */
	hipMalloc(&d_g0, nz*nx*sizeof(float));	/* gradient at previous step */
	hipMalloc(&d_g1, nz*nx*sizeof(float));	/* gradient at curret step */
	hipMalloc(&d_cg, nz*nx*sizeof(float));	/* conjugate gradient */
	hipMalloc(&d_lap, nz*nx*sizeof(float));/* laplace of the source wavefield */
	hipMalloc(&d_illum, nz*nx*sizeof(float));/* illumination of the source wavefield */
	hipMalloc(&d_pars, 4*sizeof(float));	/* d_pars[0]: obj; d_pars[1]: beta; d_pars[2]: epsilon; d_pars[3]: alpha; */
	hipMalloc(&d_alpha1, ng*sizeof(float));/* d_alpha1[]: numerator of alpha, length=ng */
	hipMalloc(&d_alpha2, ng*sizeof(float));/* d_alpha2[]: denominator of alpha, length=ng	*/
	hipMalloc(&d_vtmp, nz*nx*sizeof(float));/* temporary velocity computed with epsil */
	sf_check_gpu_error("Failed to allocate required memory!");

	/* initialize varibles */
	hipMemcpy(d_vv, vv, nz*nx*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(d_sp0, 0, nz*nx*sizeof(float));
	hipMemset(d_sp1, 0, nz*nx*sizeof(float));
	hipMemset(d_gp0, 0, nz*nx*sizeof(float));
	hipMemset(d_gp1, 0, nz*nx*sizeof(float));
	cuda_ricker_wavelet<<<(nt+511)/512,512>>>(d_wlt, amp, fm, dt, nt);
	/* configure the source/geophone geometry */
	if (!(sxbeg>=0 && szbeg>=0 && sxbeg+(ns-1)*jsx<nx1 && szbeg+(ns-1)*jsz<nz1))	
	{ sf_error("sources exceeds the computing zone!\n"); exit(1);}
	cuda_set_sg<<<(ns+511)/512,512>>>(d_sxz, sxbeg, szbeg, jsx, jsz, ns, nz);
	distx=sxbeg-gxbeg;
	distz=szbeg-gzbeg;
	if (!(gxbeg>=0 && gzbeg>=0 && gxbeg+(ng-1)*jgx<nx1 && gzbeg+(ng-1)*jgz<nz1))	
	{ sf_error("geophones exceeds the computing zone!\n"); exit(1);}
	if (csdgather)	{
		if (!(	(sxbeg+(ns-1)*jsx)+(ng-1)*jgx-distx <nx1  && (szbeg+(ns-1)*jsz)+(ng-1)*jgz-distz <nz1))	
		{ sf_error("geophones exceeds the computing zone!\n"); exit(1);}
	}
	cuda_set_sg<<<(ng+511)/512,512>>>(d_gxz, gxbeg, gzbeg, jgx, jgz, ng, nz);
	hipMemset(d_bndr, 0, nt*(2*nz+nx)*sizeof(float));
	hipMemset(d_dobs, 0, ng*nt*sizeof(float));
	hipMemset(d_dcal, 0, ng*sizeof(float));
	hipMemset(d_derr, 0, ns*ng*nt*sizeof(float));
	hipMemset(d_g0, 0, nz*nx*sizeof(float));
	hipMemset(d_g1, 0, nz*nx*sizeof(float));
	hipMemset(d_cg, 0, nz*nx*sizeof(float));
	hipMemset(d_lap, 0, nz*nx*sizeof(float));
	hipMemset(d_illum, 0, nz*nx*sizeof(float));
	hipMemset(d_pars, 0, 4*sizeof(float));
	hipMemset(d_alpha1, 0, ng*sizeof(float));
	hipMemset(d_alpha2, 0, ng*sizeof(float));
	hipMemset(d_vtmp, 0, nz*nx*sizeof(float));

	/* creat timing variables on device */
	hipEvent_t start, stop;
  	hipEventCreate(&start);	
	hipEventCreate(&stop);

	/* Our FWI is carried out via the following steps: 
		step 1. do modeling to calculate synthetic seismograms;
		step 2. backpropagate wavefield to obtain the gradient;
		step 3. compute the conjugate gradient;
		step 4. find a good step length;
		step 5. update the current velocity model with estimated step length and conjugate gradient;
		step 6. repeat step 1-5 in every iteration;
	*/
	for(iter=0; iter<niter; iter++)
	{
		hipEventRecord(start);/* record starting time */

		sf_seek(shots, 0L, SEEK_SET);
		hipMemcpy(d_g0, d_g1, nz*nx*sizeof(float), hipMemcpyDeviceToDevice);
		hipMemset(d_g1, 0, nz*nx*sizeof(float));
		hipMemset(d_illum, 0, nz*nx*sizeof(float));
		for(is=0;is<ns;is++)
		{
			/* read one observed shot record */
			sf_floatread(dobs, ng*nt, shots);
			matrix_transpose(dobs, trans, nt, ng);
			hipMemcpy(d_dobs, trans, ng*nt*sizeof(float), hipMemcpyHostToDevice);
			/* configure the source/geophone geometry */
			if (csdgather)	{
				gxbeg=sxbeg+is*jsx-distx;
				cuda_set_sg<<<(ng+511)/512, 512>>>(d_gxz, gxbeg, gzbeg, jgx, jgz, ng, nz);
			}
			hipMemset(d_sp0, 0, nz*nx*sizeof(float));
			hipMemset(d_sp1, 0, nz*nx*sizeof(float));
			/* advance time steps for source wavefield */
			for(it=0; it<nt; it++)
			{
				/* forward modeling: source */
				cuda_add_source<<<1,1>>>(d_sp1, &d_wlt[it], &d_sxz[is], 1, true);
				cuda_step_forward<<<dimg,dimb>>>(d_sp0, d_sp1, d_vv, dtz, dtx, nz, nx);
				ptr=d_sp0; d_sp0=d_sp1; d_sp1=ptr;
				/* record the calculated synthetic seismograms */
				cuda_record<<<(ng+511)/512, 512>>>(d_sp0, d_dcal, d_gxz, ng);
				cuda_cal_residuals<<<(ng+511)/512, 512>>>(d_dcal, &d_dobs[it*ng], &d_derr[is*ng*nt+it*ng], ng);
				/* saving the boundaries */
				cuda_rw_bndr<<<(2*nz+nx+511)/512,512>>>(&d_bndr[it*(2*nz+nx)], d_sp0, nz, nx, true);
			}

			hipMemset(d_gp0, 0, nz*nx*sizeof(float));
			hipMemset(d_gp1, 0, nz*nx*sizeof(float));
			/* backward propagation */
			for(it=nt-1; it>-1; it--)
			{
				/* source backpropagation */
				ptr=d_sp0;d_sp0=d_sp1;d_sp1=ptr;
				cuda_rw_bndr<<<(2*nz+nx+255)/256,256>>>(&d_bndr[it*(2*nz+nx)], d_sp1, nz, nx, false);
				cuda_step_backward<<<dimg,dimb>>>(d_illum, d_lap, d_sp0, d_sp1, d_vv, dtz, dtx, nz, nx);
				cuda_add_source<<<1,1>>>(d_sp1, &d_wlt[it], &d_sxz[is], 1, false);

				/* extrapolate residual wavefield */
				cuda_add_source<<<(ng+511)/512, 512>>>(d_gp1, &d_derr[is*ng*nt+it*ng], d_gxz, ng, true);
				cuda_step_forward<<<dimg,dimb>>>(d_gp0, d_gp1, d_vv, dtz, dtx, nz, nx);

				/* do summation for the gradient: equation 9 */
				cuda_cal_gradient<<<dimg,dimb>>>(d_g1, d_lap, d_gp1, nz, nx);
				ptr=d_gp0; d_gp0=d_gp1; d_gp1=ptr;
			}
		}
		/* calculate the value of the objective function */
		cuda_cal_objective<<<1, Block_Size>>>(&d_pars[0], d_derr, ns*ng*nt);
		hipMemcpy(&obj, &d_pars[0], sizeof(float), hipMemcpyDeviceToHost);

		/* compute source illumination and save it into snaps */
		hipMemcpy(vv, d_illum, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
		window(v0, vv, nz, nx, nz1, nx1);
		sf_floatwrite(v0, nz1*nx1, illums);

		/* compute the gradient of FWI by scaling, precondition incorporated here: equations 9 and 10 */
		cuda_scale_gradient<<<dimg,dimb>>>(d_g1, d_vv, d_illum, nz, nx, precon);
		hipMemcpy(vv, d_g1, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
		window(v0, vv, nz, nx, nz1, nx1);
		sf_floatwrite(v0, nz1*nx1, grads);
		/* Gaussian smoothing for the sharp gradient */
		cuda_bell_smoothz<<<dimg,dimb>>>(d_g1, d_illum, rbell, nz, nx);
		cuda_bell_smoothx<<<dimg,dimb>>>(d_illum, d_g1, rbell, nz, nx);

		/* calculate the factor beta in conjugate gradient method: equation 7 */
		if (iter>0) cuda_cal_beta<<<1, Block_Size>>>(&d_pars[1], d_g0, d_g1, d_cg, nz*nx); 
		hipMemcpy(&beta, &d_pars[1], sizeof(float), hipMemcpyDeviceToHost);
		/* compute the conjugate gradient */
		cuda_cal_conjgrad<<<dimg, dimb>>>(d_g1, d_cg, beta, nz, nx);
		/* estimate epsilon according to equation 11 */
		cuda_cal_epsilon<<<1, Block_Size>>>(d_vv, d_cg, &d_pars[2], nz*nx);
		hipMemcpy(&epsil, &d_pars[2], sizeof(float), hipMemcpyDeviceToHost);

		sf_seek(shots, 0L, SEEK_SET);
		hipMemset(d_alpha1, 0, ng*sizeof(float));
		hipMemset(d_alpha2, 0, ng*sizeof(float));
		/* obtain a tentative velocity model to estimate a good stepsize alpha */
		cuda_cal_vtmp<<<dimg, dimb>>>(d_vtmp, d_vv, d_cg, epsil, nz, nx);
		for(is=0;is<ns;is++)
		{
			sf_floatread(dobs, ng*nt, shots);
			matrix_transpose(dobs, trans, nt, ng);
			hipMemcpy(d_dobs, trans, ng*nt*sizeof(float), hipMemcpyHostToDevice);
			/* configure the source/geophone geometry */
			if (csdgather)	{
				gxbeg=sxbeg+is*jsx-distx;
				cuda_set_sg<<<(ng+511)/512, 512>>>(d_gxz, gxbeg, gzbeg, jgx, jgz, ng, nz);
			}
			hipMemset(d_sp0, 0, nz*nx*sizeof(float));
			hipMemset(d_sp1, 0, nz*nx*sizeof(float));
			/* remodeling with tentative velocity model vtmp */
			for(it=0; it<nt; it++)
			{
				cuda_add_source<<<1,1>>>(d_sp1, &d_wlt[it], &d_sxz[is], 1, true);
				cuda_step_forward<<<dimg,dimb>>>(d_sp0, d_sp1, d_vtmp, dtz, dtx, nz, nx);
				ptr=d_sp0; d_sp0=d_sp1; d_sp1=ptr;

				cuda_record<<<(ng+511)/512, 512>>>(d_sp0, d_dcal, d_gxz, ng);
				/* compute the numerator and the denominator of alpha: equations 5 and 12 */
				cuda_sum_alpha12<<<(ng+511)/512, 512>>>(d_alpha1, d_alpha2, d_dcal, &d_dobs[it*ng], &d_derr[is*ng*nt+it*ng], ng);
			}
		}
		/* find a good stepsize alpha: equation 5*/
		cuda_cal_alpha<<<1,Block_Size>>>(&d_pars[3], d_alpha1, d_alpha2, epsil, ng);
		hipMemcpy(&alpha, &d_pars[3], sizeof(float), hipMemcpyDeviceToHost);

		/* update the velocity model according to previous velocity, conjugate gradient and estimated stepsize */
		cuda_update_vel<<<dimg,dimb>>>(d_vv, d_cg, alpha, nz, nx);
		hipMemcpy(vv, d_vv, nz*nx*sizeof(float), hipMemcpyDeviceToHost);
		window(v0, vv, nz, nx, nz1, nx1);
		sf_floatwrite(v0, nz1*nx1, vupdates);

		hipEventRecord(stop);/* record ending time */
  		hipEventSynchronize(stop);
  		hipEventElapsedTime(&mstimer, start, stop);

		/* compute the normalized objective function */
		if(iter==0) 	{obj1=obj; objval[iter]=1.0;}
		else		objval[iter]=obj/obj1;

		if(verb) {/* output important information at each FWI iteration */
			sf_warning("obj=%f  beta=%f  epsil=%f  alpha=%f",obj, beta, epsil, alpha);
			sf_warning("iteration %d finished: %f (s)",iter+1, mstimer*1e-3);
		}
	}
	/* destroy timing varibles */
	hipEventDestroy(start);
	hipEventDestroy(stop);

	sf_floatwrite(objval,iter,objs);
	sf_fileclose(shots); 

	/* free varibles on device */
	hipFree(d_vv);
	hipFree(d_sp0);
	hipFree(d_sp1);
	hipFree(d_gp0);
	hipFree(d_gp1);
	hipFree(d_wlt);
	hipFree(d_sxz);
	hipFree(d_gxz);
	hipFree(d_bndr);
	hipFree(d_dobs);
	hipFree(d_dcal);
	hipFree(d_derr);
	hipFree(d_g0);
	hipFree(d_g1);
	hipFree(d_cg);
	hipFree(d_lap);
	hipFree(d_illum);
	hipFree(d_pars);
	hipFree(d_alpha1);
	hipFree(d_alpha2);
	hipFree(d_vtmp);
	sf_check_gpu_error("Failed to free the allocated memory!");
	/* free varibles on host */
	free(v0);
	free(vv);
	free(dobs);
	free(trans);
	free(objval);

	exit(0);
}
